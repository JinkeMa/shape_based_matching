#include "hip/hip_runtime.h"

#include<hip/hip_runtime.h>
//Ҫʹ�����ñ���blockDim�ȣ���Ҫ������ͷ�ļ�
#include<>

#include<random>

#include<opencv.hpp>

using namespace cv;


#define K 256

//ʹ��global_mem
__global__ void cu_mul(float* a,float* b,float* c)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	c[idx] = a[idx] * b[idx];
}

//ʹ��shared_mem,һάgrid,һάblock
__global__ void cu_mul2(float* a, float* b, float* c, int num_elem)
{
	__shared__ float sa[256];
	__shared__ float sb[256];
	//__shared__ float sc[256];
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < num_elem)
	{
		sa[threadIdx.x] = a[idx];
		sb[threadIdx.x] = b[idx];
	}
	
	__syncthreads();
	
	if (idx < num_elem)
	{
		c[idx] = sa[threadIdx.x] * sb[threadIdx.x];
	}
}

//transpose,ֻʹ��ȫ���ڴ�,1 elem/thread
__global__ void cu_transpose(float* a, float* b, int num_elem)
{
	//һά����
	//int idx = (gridDim.x * blockDim.x * blockIdx.y * blockDim.y + blockDim.x * blockDim.y * blockIdx.x + threadIdx.y * blockDim.x + threadIdx.x);
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (x < num_elem && y < num_elem)
	{
		b[y * num_elem + x] = a[x * num_elem + y];
	}

}

int main()
{
	//maximum 1024 threads for every block
	const int num_row = 256 * 256;
	float a[num_row], b[num_row];
	memset(b, 1.0, num_row * sizeof(float));
	//�����ڴ�
	float* A, *B, *C;
	hipMalloc(&A, num_row * sizeof(float));
	hipMalloc(&B, num_row * sizeof(float));
	hipMalloc(&C, num_row * sizeof(float));


	//��ȡ����
	cv::Mat img = cv::imread("C:/Users/user/Desktop/pic.png", 0);
	cv::resize(img, img, Size{ 256,256 });
	img.convertTo(img, CV_32F, 1.0/255,0);

	//�ƶ�����
	hipMemcpy(A, img.data, num_row * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B, b, num_row * sizeof(float), hipMemcpyHostToDevice);
	
	//32 x 32 threads/block
	dim3 block{ 32,32 };
	cu_transpose <<<64, block >>> (A, B, 256);
	//����
	hipMemcpy(b, B, num_row * sizeof(float), hipMemcpyDeviceToHost);

	cv::Mat imgb{ Size{256,256},CV_32F };
	memcpy(imgb.data, b, num_row * sizeof(float));

	std::cout << sizeof(float);

	return 1;


}